#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hiprand/hiprand.h>
#include <random>
#include <iostream>
#include <fstream>
#include <math.h>
#include <sstream>

using namespace std;

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
	printf("Error at %s:%d\n",__FILE__,__LINE__);\
	return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
	printf("Error at %s:%d\n",__FILE__,__LINE__);\
	return EXIT_FAILURE;}} while(0)

__global__ void vStep(float *v, const float *a, float *rand, int numElements) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numElements)
		rand[i] < 0.5 ? v[i] = v[i] + a[i] : v[i] = -v[i] + a[i];
}

__global__ void pStep(float *pos, const float *v, float *totalDistance,
		int numElements) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numElements) {
		pos[i] = pos[i] + v[i];
		totalDistance[i] = totalDistance[i] + abs(v[i]);
	}
}

void getRandomVector(float *vector, int n, int i) {
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, i * 12);
	hiprandGenerateUniform(gen, vector, n);
	hiprandDestroyGenerator(gen);

}

void pStepCpu(float *pos, const float *v, float *totalDistance,
		int numElements) {
	for (int i = 0; i < numElements; i++) {
		pos[i] = pos[i] + v[i];
		totalDistance[i] = totalDistance[i] + abs(v[i]);
	}
}

void vStepCpu(float *v, const float *a, int numElements) {
	for (int i = 0; i < numElements; i++) {
		rand() < 0.5 ? v[i] = v[i] + a[i] : v[i] = -v[i] + a[i];
	}
}

int main(void) {
	srand(time(NULL));
	ofstream myfile;
	int NUMBER_OF_ELEMENTS = 100;
	int ITERATIONS = 10;

	int THEARDS_PER_BLOCK = 257;
	int BLOCKS_PER_GRID = (NUMBER_OF_ELEMENTS + THEARDS_PER_BLOCK - 1)
			/ THEARDS_PER_BLOCK;

	std::stringstream sstm;
	sstm << "simulation_it_" << ITERATIONS << "_NOE_" << NUMBER_OF_ELEMENTS;
	string name = sstm.str();
	myfile.open(name);

	hipEvent_t start_alloc, stop_alloc, start_computing_GPU,
			stop_computing_GPU, start_computing_CPU, stop_computing_CPU,
			start_copyback, stop_copyback;

	hipEventCreate(&start_alloc);
	hipEventCreate(&stop_alloc);
	hipEventCreate(&start_computing_GPU);
	hipEventCreate(&stop_computing_GPU);
	hipEventCreate(&start_computing_CPU);
	hipEventCreate(&stop_computing_CPU);
	hipEventCreate(&start_copyback);
	hipEventCreate(&stop_copyback);

	size_t size = NUMBER_OF_ELEMENTS * sizeof(float);
	float milliseconds;

	float *posX = (float *) malloc(size), *posY = (float *) malloc(size), *vY =
			(float *) malloc(size), *vX = (float *) malloc(size), *aX =
			(float *) malloc(size), *aY = (float *) malloc(size),
			*totalDistanceX = (float *) malloc(size), *totalDistanceY =
					(float *) malloc(size);

	// initial values of velocity vector
	default_random_engine generator;
	normal_distribution<double> distribution(1, 2.5);

	for (int i = 0; i < NUMBER_OF_ELEMENTS; i++) {
		vX[i] = distribution(generator);
		vY[i] = distribution(generator);
	}

	// initial values of acceleration vector
	for (int i = 0; i < NUMBER_OF_ELEMENTS; i++) {
		double lenght = sqrt(vX[i] * vX[i] + vY[i] * vY[i]);
		aX[i] = (vX[i] / lenght) * 0.025;
		aY[i] = (vY[i] / lenght) * 0.025;
	}

	// initial values of position vector also setting the total distance as zero
	for (int i = 0; i < NUMBER_OF_ELEMENTS; i++) {
		posX[i] = 0;
		posY[i] = 0;
		totalDistanceY[i] = 0;
		totalDistanceX[i] = 0;
	}

	// allocating memory on device
	hipEventRecord(start_alloc);
	float *posXd = NULL, *posYd = NULL, *vYd = NULL, *vXd = NULL, *aXd = NULL,
			*aYd = NULL, *randomd = NULL, *totalDistanceXd = NULL,
			*totalDistanceYd = NULL;

	hipMalloc((void **) &posXd, size);
	hipMalloc((void **) &posYd, size);
	hipMalloc((void **) &vYd, size);
	hipMalloc((void **) &vXd, size);
	hipMalloc((void **) &aXd, size);
	hipMalloc((void **) &aYd, size);
	hipMalloc((void **) &randomd, size);
	hipMalloc((void **) &totalDistanceXd, size);
	hipMalloc((void **) &totalDistanceYd, size);

	// copying the initial values on device
	hipMemcpy(posXd, posX, size, hipMemcpyHostToDevice);
	hipMemcpy(posYd, posY, size, hipMemcpyHostToDevice);
	hipMemcpy(vYd, vY, size, hipMemcpyHostToDevice);
	hipMemcpy(vXd, vX, size, hipMemcpyHostToDevice);
	hipMemcpy(aXd, aX, size, hipMemcpyHostToDevice);
	hipMemcpy(aYd, aY, size, hipMemcpyHostToDevice);
	hipMemcpy(totalDistanceXd, totalDistanceX, size, hipMemcpyHostToDevice);
	hipMemcpy(totalDistanceYd, totalDistanceY, size, hipMemcpyHostToDevice);

	hipEventRecord(stop_alloc);
	hipEventSynchronize(stop_alloc);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_alloc, stop_alloc);
	myfile
			<< "% ---------------------------------------------------------- %\n";
	myfile << "allocation_time = " << milliseconds << ";\n";

	// simulation CPU
	hipEventRecord(start_computing_CPU);
	for (int i = 0; i < ITERATIONS; i++) {
		vStepCpu(vX, aX, NUMBER_OF_ELEMENTS);
		vStepCpu(vY, aY, NUMBER_OF_ELEMENTS);
		pStepCpu(vX, aX, totalDistanceX, NUMBER_OF_ELEMENTS);
		pStepCpu(vY, aY, totalDistanceY, NUMBER_OF_ELEMENTS);
	}
	hipEventRecord(stop_computing_CPU);
	hipEventSynchronize(stop_computing_CPU);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_computing_CPU,
			stop_computing_CPU);

	// all that matters to me is execution time
	myfile << "cpu_compution_time = " << milliseconds << ";\n";

	// simulation GPU
	hipEventRecord(start_computing_GPU);
	for (int i = 1; i < ITERATIONS; i++) {
		getRandomVector(randomd, NUMBER_OF_ELEMENTS, i);
		vStep<<<BLOCKS_PER_GRID, THEARDS_PER_BLOCK>>>(vXd, aXd, randomd, NUMBER_OF_ELEMENTS);
		vStep<<<BLOCKS_PER_GRID, THEARDS_PER_BLOCK>>>(vYd, aYd, randomd, NUMBER_OF_ELEMENTS);
		pStep<<<BLOCKS_PER_GRID, THEARDS_PER_BLOCK>>>(posXd,vXd,totalDistanceXd, NUMBER_OF_ELEMENTS);
		pStep<<<BLOCKS_PER_GRID, THEARDS_PER_BLOCK>>>(posYd,vYd,totalDistanceYd, NUMBER_OF_ELEMENTS);
	}

	hipEventRecord(stop_computing_GPU);
	hipEventSynchronize(stop_computing_GPU);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_computing_GPU,
			stop_computing_GPU);

	myfile << "gpu_compution_time = " << milliseconds << ";\n";

	// getting the results back to host
	hipEventRecord(start_copyback);
	hipMemcpy(posX, posXd, size, hipMemcpyDeviceToHost);
	hipMemcpy(posY, posYd, size, hipMemcpyDeviceToHost);
	hipMemcpy(totalDistanceX, totalDistanceXd, size, hipMemcpyDeviceToHost);
	hipMemcpy(totalDistanceY, totalDistanceYd, size, hipMemcpyDeviceToHost);

	hipEventRecord(stop_copyback);
	hipEventSynchronize(stop_copyback);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, stop_copyback, stop_copyback);

	myfile << "copyback_time = " << milliseconds << ";\n";

	// print section
	myfile
			<< "% ---------------------------------------------------------- %\n";
	myfile << "number_of_particles = " << NUMBER_OF_ELEMENTS << ";\n";
	myfile << "number_of_iterations = " << ITERATIONS << ";\n";
	myfile
			<< "% ---------------------------------------------------------- %\n";

	myfile << "displacement = [";
	for (int i = 0; i < NUMBER_OF_ELEMENTS - 1; i++) {
		myfile << sqrt(posX[i] * posX[i] + posY[i] * posY[i]) << ", ";
	}
	myfile
			<< sqrt(
					posX[NUMBER_OF_ELEMENTS - 1] * posX[NUMBER_OF_ELEMENTS - 1]
							+ posY[NUMBER_OF_ELEMENTS - 1]
									* posY[NUMBER_OF_ELEMENTS - 1]) << "];\n";

	myfile << "final_positions = [";
	for (int i = 0; i < NUMBER_OF_ELEMENTS - 1; i++) {
		myfile << "[" << posX[i] << ", " << posY[i] << "], ";
	}
	myfile << "[" << posX[NUMBER_OF_ELEMENTS - 1] << ", "
			<< posY[NUMBER_OF_ELEMENTS - 1] << "]];\n";

	myfile << "total_distances = [";
	for (int i = 0; i < NUMBER_OF_ELEMENTS - 1; i++) {
		myfile << sqrt(totalDistanceX[i] + totalDistanceY[i]) << ", ";
	}
	myfile
			<< sqrt(
					totalDistanceX[NUMBER_OF_ELEMENTS - 1]
							+ totalDistanceY[NUMBER_OF_ELEMENTS - 1]) << "];\n";

	myfile << "final_velocities = [";
	for (int i = 0; i < NUMBER_OF_ELEMENTS - 1; i++) {
		myfile << sqrt(vX[i] * vX[i] + vY[i] * vY[i]) << ", ";
	}
	myfile
			<< sqrt(
					vX[NUMBER_OF_ELEMENTS - 1] * vX[NUMBER_OF_ELEMENTS - 1]
							+ vY[NUMBER_OF_ELEMENTS - 1]
									* vY[NUMBER_OF_ELEMENTS - 1]) << "];\n";

	myfile.close();

	// free the allocated memory
	free(posX);
	free(posY);
	free(vY);
	free(vX);
	free(aX);
	free(aY);
	free(totalDistanceX);
	free(totalDistanceY);

	hipFree(posXd);
	hipFree(posYd);
	hipFree(vYd);
	hipFree(vXd);
	hipFree(aXd);
	hipFree(aYd);
	hipFree(totalDistanceXd);
	hipFree(totalDistanceYd);

	return 0;
}

